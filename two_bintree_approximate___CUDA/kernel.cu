#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdbool.h>
#include <setjmp.h>


#define TREE_space_extra_buff 4
#define point_space_extra_buff 8
#define COUNT 20

#define DATASET_NUM 9
#define MAX_INT_DEF 0xfffffff
#define max_clock_stamp 0xfffffff
#define max_clock_store 16
#define DIM 3
jmp_buf jmpbuffer;
clock_t run_time_debug[max_clock_store];
int clock_index=0;
bool write_Data_head = false;
unsigned long long  store_avi_node_num=0;
typedef struct point{
    float values[DIM];
    float th;//store distance or quantity
}point;
typedef struct node{
    point data;
    struct node* left=NULL;
    struct node* right=NULL;
}node;
unsigned long long mypow(int x, int y) {
    unsigned long long result = 1;
    int i;
    for (i = 0; i < y; i++) {
        result *= x;
    }
    //    printf("%llu",result);
    return result;
}
void print_nD_arr(point* arr){
    int size=(int)roundf(arr[0].th);//roundf: in order to make sure floating error does not affect size number!
    printf("index");
    for (int k = 0; k <DIM ; ++k) printf("\t\tdata[%d]\t",k);
    printf("\t\tth/dist\n");

    for (int i = 0; i <=size ; ++i) {
        printf("%d\t\t",i);
        for (int j = 0; j <DIM ; ++j) {
            printf("%f\t\t",arr[i].values[j]);
        }
        printf("%d\n",(int)arr[i].th);
    }
}
void print_this_point_woth(point thispoint){
    printf("(");
    for (int i = 0; i <DIM ; ++i) printf("%.1f, ",thispoint.values[i]);
    printf("\b\b)");
}
__device__
void print_this_point_woth_gpu(point thispoint) {
    printf("(");
    for (int i = 0; i < DIM; ++i) printf("%.1f, ", thispoint.values[i]);
    printf("\b\b)");
}
void print_this_point(point thispoint){
    printf("(");
    for (int i = 0; i <DIM ; ++i) printf("%.1f, ",thispoint.values[i]);
    printf("\b\b)_");
    printf("%.1f",thispoint.th);
    printf("\n");
}
void swap(point *x,point *y){
    point tmp;
    tmp=*x;
    *x=*y;
    *y=tmp;
}
point *super_gen_seq_arr(int number,bool reversed){
    int i,dim,j;
    point *arr=(point*)malloc(sizeof(point)*(number+1));
    for (i = 1; i <=number ; i++) {
        j=i;
        if(reversed==true) {
            j=number-i+1;
        }
        for (dim = 0; dim <DIM ; dim++) {
            arr[i].values[dim]=(float)(dim*100+j);//init
        }
        arr[i].th=MAX_INT_DEF;//init
    }
    arr[0].th=number;
    return arr;
}
point *super_gen_rand_arr(int number,int max){
    srand(time(NULL));
    int i,dim;
    point *arr=(point*)malloc(sizeof(point)*(number+1));
    for (i = 1; i <=number ; i++) {
        for (dim = 0; dim <DIM ; dim++) {
            arr[i].values[dim]=(float)(rand() % (max+1));//should use this

//            arr[i].values[dim]=((float)(rand()%100));//init
        }
        arr[i].th=MAX_INT_DEF;//init
    }
    arr[0].th=number;
    return arr;
}
point* deep_copy(point *arr){
    int size=(int)roundf(arr[0].th);
    point* newarr=(point*)malloc(sizeof(point)*(size+1));
    memcpy(newarr,arr, sizeof(point)*(size+1));
    return newarr;
}
int print_test_qsort(point* arr){
    int val=0;
    for (int i = 1; i <=(int)roundf(arr[0].th) ; ++i) {
        val+=mypow(10,(int)roundf(arr[0].th)-i)*(int)arr[i].values[0];
    }
    return val;
}
void quicksort(point *orgarr,int first,int last,int for_which_dim){
    int from_first,from_last,pivot;
//    int testing;
//    int test_from_first_val;
//    int test_from_last_val;
//    int test_pivot_val;
//    testing=print_test_qsort(orgarr);
    if(for_which_dim>DIM){
        printf("dim Err into quick sort\n");
        EXIT_FAILURE;
    }
    if(first<last){
        pivot=first;
        from_first=first;
        from_last=last;
        while(from_first<from_last){//if left index & right index not cross mid-> continue
            //if not normal-> move the index
            while((orgarr[from_first].values[for_which_dim]<=orgarr[pivot].values[for_which_dim])&&(from_first<last)) from_first++;
            //if not normal-> move the index
            while(orgarr[from_last].values[for_which_dim]>orgarr[pivot].values[for_which_dim]) from_last--;
            //            //if valid first and last index-> swap two chosen points (1 at right and another ar left)
            if(from_first<from_last)    swap(&orgarr[from_first],&orgarr[from_last]);
//            otherwise continue
//            printf("----\n");
//            print_nD_arr(orgarr);
//            usleep(1000000*1);
//            print_nD_arr(orgarr);
        }
        //change the pivot to the right side of the chosen point
        swap(&orgarr[pivot],&orgarr[from_last]);
        //insert node for right side of the tree
        quicksort(orgarr,first,from_last-1,for_which_dim);
        //insert node for left side of the tree
        quicksort(orgarr,from_last+1,last,for_which_dim);
    }
}
void print2DUtil(node *root, int space){
    if (root == NULL) return;
    int i;
    space += COUNT;

    print2DUtil(root->right, space);
    printf("\n");
    for (i = COUNT; i < space; i++) printf(" ");
    printf("(");
    for (i = 0; i <DIM ; i++) {
        printf("%.1f  ",root->data.values[i]);
    }
    printf(")\n");
//    printf("(%d,%d)\n", root->data.,root->data.y);
    print2DUtil(root->left, space);
}
void print_node(node* root){
    int i;

    printf("(");
    for (i = 0; i <DIM ; i++) {
        printf("%.1f ",root->data.values[i]);
    }
    printf(")th:%d\n",(int)roundf(root->data.th));
}
int print_bt(node* root){
    static int count=0;
    int i;
    if(root==NULL) return 0;
//    usleep(0.1*1000000);
    printf("(");
    for (i = 0; i <DIM ; i++) {
        printf("%.1f ",root->data.values[i]);
    }
    printf(")th:%d\n",(int)roundf(root->data.th));
    count++;
    print_bt(root->left);
    print_bt(root->right);
    return count;
}
int super_rand(int min,int max){
    srand(time(NULL));
    return rand()%(max+1-min)+min;
}
int find_mid_index(point* sorted_arr,point target,int chosen_dim){
    int i;
    for(i=1;i<=(int)sorted_arr[0].th;i++){
        if(sorted_arr[i].values[chosen_dim]>=target.values[chosen_dim]) return i-1;//previous index
    }
}
void show_time(char* str){
    run_time_debug[clock_index%max_clock_store]=clock();
    printf("%d____%d ms__ %s\n",clock_index,(int)(1000*(run_time_debug[clock_index%max_clock_store]-run_time_debug[((clock_index-1))%max_clock_store])/CLOCKS_PER_SEC),str);
    if(clock_index==max_clock_stamp) exit(10);
    clock_index++;
}
point* super_selection(point *orgarr,const char *up_down,int choose_dim,bool random_pick_med){
//    int portion=100/split_portion;// for annoy should change here! maybe: int->float//original
    // for GPU. Generate 32 kinds of portion
    //printf("--------new super selection----------\n");//orgprint

    int orgsorted_size=(int)roundf(orgarr[0].th);
    point *new_arr;
    int new_arr_size;
    int i;
    int mid_index;
    point mid_point;
    //orginial_arr_size is same as sorted_arr_size
//    show_time("initialize super_selection");
    point *sorted_orgarr=deep_copy(orgarr);
    quicksort(sorted_orgarr,1,orgsorted_size,choose_dim);
//    show_time("Quick sort");
//    if(orgarr[0].th<=3) random_pick_med=false;

    if (random_pick_med==true && (int)sorted_orgarr[0].th>1){
        //rand pick 2 points and calc the mean with random only pick an index with randonly pick 2 index
        int rindex1,rindex2;//index1 & index2
        point val1,val2;
        rindex1=super_rand(1,(int)sorted_orgarr[0].th);
//        show_time("find rindex1");
        do{
            rindex2=super_rand(1,(int)sorted_orgarr[0].th);
            if(rindex1==rindex2) rindex2= (rindex2+super_rand(0,(int)sorted_orgarr[0].th-2))%(int)sorted_orgarr[0].th+1;
                //org rindex2+- rand()
        }while(rindex1==rindex2);//randomed value cannot be the same//but condition variable is slow So this is just a backup plan
//        show_time("find rindex2");
        //printf("index=%d,%d\n",rindex1,rindex2);//orgprint
        //calc where should the index should be inserted in the array
        val1=sorted_orgarr[rindex1];
        val2=sorted_orgarr[rindex2];
        //find out the mid value
        for(i=0;i<DIM;i++) mid_point.values[i]=(val1.values[i]+val2.values[i])/2;//ignore the th value//FUTURE: can be simplify to one dim only
//        show_time("find virtual point");
        //find out the cutting index--with dim
        mid_index=find_mid_index(sorted_orgarr,mid_point,choose_dim);

        //printf("using %.1f as mid_index\n",((float)mid_index+0.5));//orgprint
//        show_time("find mid_index");

    }else if(!random_pick_med && (int)sorted_orgarr[0].th>1){
        mid_index = (int) ((1 + orgsorted_size) / 2);
        for(i=0;i<DIM;i++) mid_point.values[i]=((sorted_orgarr[mid_index].values[i]+sorted_orgarr[mid_index+1].values[i])/2);//deleted one or previous one
    }else if((int)sorted_orgarr[0].th<=1){
        new_arr=(point*)malloc(sizeof(point));//one point array
        new_arr[0].th=0;
        for(i=0;i<DIM;i++) new_arr[0].values[i]=sorted_orgarr[1].values[i];//deleted one or previous one
//        show_time("Edge-- End of leaf");
        return new_arr;
    }
    //for when only 1 element left
//    show_time("figure out mid_point & mid_split");
    if(strcmp(up_down,"down")==0){
//        printf("DOWN\n");
        new_arr_size=mid_index;
        new_arr=(point*)malloc(sizeof(point)*(1+new_arr_size));
        for(i=1;i<=new_arr_size;i++) new_arr[i]=sorted_orgarr[i];
        for(i=0;i<DIM;i++) new_arr[0].values[i]=mid_point.values[i];
//        show_time("down arr created!");
    }else if(strcmp(up_down,"up")==0){
//        printf("UP\n");
        new_arr_size=orgsorted_size-mid_index;// for annoy should change here!
        new_arr=(point*)malloc(sizeof(point)*(1+new_arr_size));
        for(i=1;i<=new_arr_size;i++) new_arr[i]=sorted_orgarr[mid_index+i];
        for(i=0;i<DIM;i++) new_arr[0].values[i]=mid_point.values[i];
//        show_time("up arr created!");
    }else{
        printf("Debug: arr is empty & super_selection failed!!!\n");
        exit(0);
    }

    new_arr[0].th=(float)new_arr_size;
    return new_arr;
}
node* convert_2_KDtree_code(point* arr, node* new_nodes, unsigned long long* node_index, float th, int brute_force_range, int chosen_dim, bool random_med) {
    if (new_nodes == NULL) {
        //handle error and rerun
        printf("Err handled!\n");
        longjmp(jmpbuffer, 1);
    }
    unsigned long long index_stamp = *node_index;
    //if (index_stamp==0) {
    //    printf("index_stamp%d \t\tstore_avi_node_num%d\n", index_stamp, store_avi_node_num);
    //}
    if (index_stamp > store_avi_node_num || index_stamp<0) {
        printf("index_stamp%d > store_avi_node_num%d\n", index_stamp, store_avi_node_num);
        longjmp(jmpbuffer, 2);
    }
    //    node* new_node=(node*)malloc(sizeof(node));
    point* arr_left;//=(point*) malloc(sizeof(point)*(arr[0].th+1));
    point* arr_right;//=(point*) malloc(sizeof(point)*(arr[0].th+1));
    int i;
    //    printf("\nEach recusrsion array\n");
    //    print_nD_arr(arr);
    chosen_dim++;
    chosen_dim %= DIM;
    //printf("Current Dim %d____node_index %d\n", chosen_dim, *node_index);//orgprint
    //    printf("updown st\n");
    arr_left = (super_selection(arr, "down", chosen_dim, random_med));//too slow!!!!!!!!!!!fix here----fixed!
    arr_right = (super_selection(arr, "up", chosen_dim, random_med));
    //    printf("updown End\n");
    //handle error
    new_nodes[index_stamp].data.th = th;
    if ((int)roundf(arr_left[0].th) >= brute_force_range) {
        for (i = 0; i < DIM; i++) new_nodes[index_stamp].data.values[i] = arr_left[0].values[i];
        //printf("L\n");//orgprint
        //print_nD_arr(arr_left);//orgprint
        //print_node(&new_nodes[index_stamp]);//orgprint
        (*node_index)++;
        new_nodes[index_stamp].left = convert_2_KDtree_code(arr_left, new_nodes, &(*node_index), th, brute_force_range, chosen_dim, random_med);
        free(arr_left);
    }
    else {
        for (i = 0; i < DIM; i++) new_nodes[index_stamp].data.values[i] = arr_left[0].values[i];
        //printf("L----NULL\n");////orgprint
        //print_nD_arr(arr_left);////orgprint
        //print_node(&new_nodes[index_stamp]);//orgprint
        new_nodes[index_stamp].left = NULL;
        free(arr_left);
    }
    if ((int)roundf(arr_right[0].th) >= brute_force_range) {
        for (i = 0; i < DIM; i++) new_nodes[index_stamp].data.values[i] = arr_right[0].values[i];
        //printf("R\n");////orgprint
        //print_nD_arr(arr_right);//orgprint
        //print_node(&new_nodes[index_stamp]);//orgprint
        (*node_index)++;
        new_nodes[index_stamp].right = convert_2_KDtree_code(arr_right, new_nodes, &(*node_index), th, brute_force_range, chosen_dim, random_med);
        free(arr_right);
    }
    else {
        for (i = 0; i < DIM; i++) new_nodes[index_stamp].data.values[i] = arr_right[0].values[i];
        //printf("R----NULL\n");//orgprint
        //print_nD_arr(arr_right);//orgprint
        //print_node(&new_nodes[index_stamp]);//orgprint
        new_nodes[index_stamp].right = NULL;
        free(arr_right);
    }
    //printf("------------------pop------------------------");//orgprint
    //printf("index_stamp: %d\n", index_stamp);//orgprint
    //if (index_stamp == 6) {};//debug only
    return &new_nodes[index_stamp];
}
int log2n(unsigned int n) {
    return (n > 1) ? 1 + log2n(n / 2) : 0;
}
unsigned long long calc_total_node_number(point* arr) {
    //    print_nD_arr(arr);
    unsigned long long buff;
    double log2x = log2n((int)arr[0].th);
    buff = pow(2, (int)((log2x)+1));
    
   
    //return 2 * buff + TREE_space_extra_buff;
    return buff - 1 + (int)arr[0].th + TREE_space_extra_buff;//normally the TREE_space_extra_buff should be zero!
//    return 2*(buff)-1-(buff-(int)arr[0].th)+TREE_space_extra_buff;//normally the TREE_space_extra_buff should be zero!
}
node* convert_2_KDtree(point* arr, bool random_med,unsigned long long rectify) {
    node* new_nodes;
    unsigned long long node_num, node_index;
    node_num = calc_total_node_number(arr)+rectify;
    //re-modify node_num
    store_avi_node_num = node_num;
    //printf("the node_number is: %d\n", node_num);//orgprint
    //new_nodes = (node*)malloc(sizeof(node) * node_num);
    hipMallocManaged(&new_nodes, sizeof(node) * node_num);
    node_index = 0;

    return convert_2_KDtree_code(arr, new_nodes, &node_index, 1, 1, -1, random_med);
}
void push_front(point* org_arr,point desire_push,int k,bool k_full_lock){//k_full_lock: true to avoid element be popped if queue overflow!
//    printf("----------------------------------------------------\n");
    //need to update the arr[0].th as well!
    if(k_full_lock && k<=org_arr[0].th) return;
    int i;
    org_arr[0].th+=(float)(1-(int)(k<=(int)org_arr[0].th));
//    printf("%d\n",org_arr[0].th);
    for (i = (int)roundf(org_arr[0].th); i>1 ; i--) {
//        printf(" %d",i);
        org_arr[i]=org_arr[i-1];
    }
//    printf("\n");
    org_arr[1]=desire_push;
//    return org_arr;
}
void push_back(point* org_arr,point desire_push,int k, bool k_full_lock){//k_full_lock: true to avoid element be popped if queue overflow!
    if(k_full_lock && k<=(int)org_arr[0].th) return;
    int i;
    if(k<=(int)org_arr[0].th){
        for(i=1;i<(int)org_arr[0].th;i++) org_arr[i]=org_arr[i+1];
    }
    org_arr[0].th+=(float)(1-(int)(k<=(int)org_arr[0].th));
    org_arr[(int)org_arr[0].th]=desire_push;

}
void distance_calc(point target, point *on_leaf){
    double dist=0;
    int dim;
    for (dim = 0; dim <DIM ; dim++) {
        dist+= pow(target.values[dim]-on_leaf->values[dim],2);
    }
    dist=pow(dist,0.5);
    on_leaf->th=(float)dist;
//    return (float) dist;
}
__device__
void distance_calc_gpu(point target, point* on_leaf) {
    double dist = 0;
    int dim;
    for (dim = 0; dim < DIM; dim++) {
        dist += pow(target.values[dim] - on_leaf->values[dim], 2);
    }
    dist = pow(dist, 0.5);
    on_leaf->th = (float)dist;
    //    return (float) dist;
}
void k_nearest_search_code(int k,node* root,bool approximate,point target,int chosen_dim,point* nearest_points){
    //under occasion: approximate==true && only one point
    // this recursion is for approximate kNN search where k=1
    if(nearest_points[0].th>=(float)k) return;//return when have found k's element
    if(approximate) {
        if (root == NULL) return;
        else printf("--->%.1f", root->data.values[chosen_dim]);
        bool is_leaf = (root->left == NULL) && (root->right == NULL);
        if ((nearest_points[1].values[chosen_dim] != root->data.values[chosen_dim] || nearest_points[0].th == 0) &&
            (is_leaf)) {//(value comapre|| init)&&(is leaf)
            printf("S\t");//S means store!
            distance_calc(target, &root->data);
//            push_front(nearest_points, root->data, k,true);
            push_back(nearest_points, root->data, k,true);
        }//need modified when k>1
        if (target.values[chosen_dim] < root->data.values[chosen_dim]) {
            chosen_dim++;
            chosen_dim %= DIM;
            k_nearest_search_code(k, root->left, approximate, target, chosen_dim, nearest_points);
        } else {
            chosen_dim++;
            chosen_dim %= DIM;
            k_nearest_search_code(k, root->right, approximate, target, chosen_dim, nearest_points);
        }
    }else {
        if(chosen_dim==0) printf("\n");//for printing
        if (root == NULL) return;
        else{printf("----->(");for (int i = 0; i <DIM ; ++i) {printf("%.1f ", root->data.values[i]); }printf(")");}
        bool is_leaf = (root->left == NULL) && (root->right == NULL);
        if ((nearest_points[1].values[chosen_dim] != root->data.values[chosen_dim] || nearest_points[0].th == 0) && (is_leaf)) {//(value comapre|| init)&&(is leaf)
            printf("S\t");//S means store!
            distance_calc(target, &root->data);
//            push_front(nearest_points, root->data, k,true);
            push_back(nearest_points, root->data, k,true);
        }//need modified when k>1
        if (target.values[chosen_dim] < root->data.values[chosen_dim]) {
            chosen_dim++;
            chosen_dim %= DIM;
            k_nearest_search_code(k, root->left, approximate, target, chosen_dim, nearest_points);
            if(root->right!=NULL){
                k_nearest_search_code(k, root->right, approximate, target, chosen_dim, nearest_points);
            }
        } else {
            chosen_dim++;
            chosen_dim %= DIM;
            k_nearest_search_code(k, root->right, approximate, target, chosen_dim, nearest_points);
            if(root->left!=NULL){
                k_nearest_search_code(k, root->left, approximate, target, chosen_dim, nearest_points);
            }
        }
    }
}
point* k_nearest_search(int k,node* tree,bool approximate,point target){
    point* nearest_points=(point*)malloc(sizeof(point)*(k+1));
    nearest_points[0].th=0;
    printf("value searched(S:stored): ");
    k_nearest_search_code(k,tree,approximate,target,0,nearest_points);
    printf("\n");
    return nearest_points;
}
point k_nearest_search_wo_recursion_stack_k1_approx_code(node* root,point target){//return nearest_point
    node* current=root;
    int dim_count=0;
    printf("traverse route:");
    print_this_point_woth(current->data);
    while(current->left && current->right){
        if(current->data.values[dim_count]>target.values[dim_count] && current->left){
            current=current->left;
        } else if (current->data.values[dim_count]<=target.values[dim_count] && current->right){
            current=current->right;
        }
        else{
            if(current->left) current=current->left;
            else current=current->right;
        }
        dim_count++;
        dim_count%=DIM;
        printf("--->");
        print_this_point_woth(current->data);
        if(dim_count==0) printf("\n");
    }
    distance_calc(target,&current->data);
    printf("\n");
    return current->data;
}
__device__
void test_gpu_cancompile() {
    printf("Can!!!\n");
}
__global__ 
void k_nearest_search_k1_GPU(node** root,point target,int tree_num,point* point_list){//return nearest_point
    
    int i;// , route_store;
    int dim_count;
    //int debug;

    i = blockIdx.x * blockDim.x + threadIdx.x;
   // printf("\n----------------------thread------------------------------------%d\n", i);
    if(i<tree_num){
        if (root[i] == NULL) {
            printf("Null root skip!\n");
            return;
        }
        //printf("thread(%d) working!\n", i);//orgprint
        dim_count=0;
        while(root[i]->left && root[i]->right){
            if(root[i]->data.values[dim_count]>target.values[dim_count] && root[i]->left){
                root[i] = root[i]->left;
            } else if (root[i]->data.values[dim_count]<=target.values[dim_count] && root[i]->right){
                root[i] = root[i]->right;
            }
            else{
                if(root[i]->left) root[i] = root[i]->left;
                else root[i] = root[i]->right;
            }
            dim_count++;
            dim_count%=DIM;
        }
        distance_calc_gpu(target,&root[i]->data);
        point_list[i]= root[i]->data;
        /*debug = 4;*/
    }
}
point* k_nearest_search_wo_recusrion_stack(int k,node* tree,bool approximate,point target){
    point* nearest_points=(point*)malloc(sizeof(point)*(k+1));
    nearest_points[0].th=(float)k;
    int i;
    point onenearest;
    if(k==1 && approximate) {
        onenearest=k_nearest_search_wo_recursion_stack_k1_approx_code(tree,target);
//        print_this_point(onenearest);
        nearest_points[1]=onenearest;
    }
    else{
        printf("Sorry not yet finish this part yet!\n");
    }
    show_time("found the nearest point(s)");
    return nearest_points;
}
int gpu_kd_portion(int parallel_num,int scaling){//scaling=1~parallel_num
    return parallel_num/scaling;
}
void write_data_to_txt(char* fname,point* arr){
    FILE *f=fopen(fname,"w");
    if(f==NULL) exit(2);
    int k,i;

    fprintf_s(f,"%d %d\n",DIM,(int)arr[0].th);
    for (i = 1; i <=(int)arr[0].th ; ++i) {
//        fprintf_s(f,"%d\t\t",i);
        for (int j = 0; j <DIM ; ++j) {
            fprintf_s(f,"%f\t\t",arr[i].values[j]);
        }
        fprintf_s(f,"%d\n",(int)arr[i].th);
    }
    fclose(f);
}
point* read_data_from_txt(char* fname){
    FILE *f;
    char *orgarr;//have to mind the dataset length!!
    f=fopen(fname,"r+");
    if(f==NULL) exit(2);
    int dim,num_data,i,j;
    fscanf(f,"%d %d\n",&dim,&num_data);//second line to read info
    printf("dim:%d\tdatanum:%d\n",dim,num_data);
//    float buffdata[num_data+1][dim];
//    int buffth[num_data];
    point *input;
    input=(point*)malloc(sizeof(point)*(num_data+1));
    for(i=1;i<=num_data;i++){
        //perline
        for(j=0;j<dim;j++) fscanf(f,"%f\t\t",&input[i].values[j]);
        fscanf(f,"%f\n",&input[i].th);
    }
    fclose(f);
    input[0].th=(float)num_data;
    return input;
}
int calc_node_rounte_space_avg(int data_num) {
    return ((log(data_num) / log(2) + 1) + data_num) / 2;
}
void write_traverseresult_to_disk(char rn,int num_tree, unsigned long long num_queries, double time_taken, unsigned long long queries_max, unsigned long long  queries_min, unsigned long long  queries_interval, int block_num, int threads_num_per_block) {
    FILE* file;
    char buffer[128];
    char tmp[16];
    char fname_format[] = "%cblock_num%d,threads_per_block%d,treenum%d,quiers_max%llu,quiers_min%llu,quiers_interval%llu .txt";
    char fname[sizeof fname_format + 128];
    sprintf(fname, fname_format,rn,block_num,threads_num_per_block, num_tree, queries_max,queries_min,queries_interval);

    if (write_Data_head) {
        file = fopen(fname, "a");
        fprintf_s(file, " %d |  %llu |  %lf\n", num_tree, num_queries, time_taken);
    }
    else {
        write_Data_head = true;
        file = fopen(fname, "w");
        fprintf_s(file, "num_tree | num_queries | time_taken(s)\n");
    }
    /*//num_tree
    strcpy(buffer, "num_tree: ");
    itoa(num_tree, tmp, 10);
    strcat(buffer, tmp);
    
    //num_queries
    strcat(buffer, "  |  num_queries");
    ulltoa(num_queries, tmp, 10);
    strcat(buffer, tmp);

    //time_taken
    strcat(buffer, "  |  num_queries");
    sprintf(tmp, "%lf", time_taken);
    strcat(buffer, tmp);
    fprintf(file, "%s\n", buffer);*/

    fclose(file);
}
int main(){
    printf("process starts!\n");
    clock_t main_start;
    run_time_debug[0]=main_start=clock();
//    point* orgarr;
//    orgarr=super_gen_seq_arr(DATASET_NUM,true);
//    orgarr=super_gen_rand_arr(DATASET_NUM,48);
//    print_nD_arr(orgarr);//print!

//    test deepcopy--successful
/*
 * arr2=orgarr;//link
    arr2=deep_copy(orgarr);//deep copy
    arr2[0].values[0]=99999;
    print_nD_arr(orgarr);
 */
//    test swap & quick sort
/*

//    point* testarr=super_gen_seq_arr(7,true);
    point* testarr=super_gen_rand_arr(21);
//    testarr[0].values[0]=99999;testarr[0].values[1]=99999;
    print_nD_arr(testarr);
//    swap(&testarr[3],&testarr[6]);
    quicksort(testarr,1,21,2);
    printf("End\n");
    print_nD_arr(testarr);
*/
//test super_selection
/*

    printf("\n------------------------------------------------------------------\n");
    point* qsarr=deep_copy(orgarr);quicksort(qsarr,1,DATASET_NUM,0);print_nD_arr(qsarr);
    qsarr=deep_copy(orgarr);quicksort(qsarr,1,DATASET_NUM,1);print_nD_arr(qsarr);
    qsarr=deep_copy(orgarr);quicksort(qsarr,1,DATASET_NUM,2);print_nD_arr(qsarr);

    printf("\n------------------------------------------------------------------\n");
    print_nD_arr(super_selection(orgarr,"down",0,50));//print_nD_arr(selected);
    print_nD_arr(super_selection(orgarr,"down",1,50));//print_nD_arr(selected);
    print_nD_arr(super_selection(orgarr,"down",2,50));//print_nD_arr(selected);
    printf("\n------------------------------------------------------------------\n");
    print_nD_arr(super_selection(orgarr,"up",0,50));//print_nD_arr(selected);
    print_nD_arr(super_selection(orgarr,"up",1,50));//print_nD_arr(selected);
    print_nD_arr(super_selection(orgarr,"up",2,50));//print_nD_arr(selected);
*/
//test push
/*

    printf("------------test push\n");
    point target={{51,32,61},0};
    point target1={{1,32,61},0};
    point* org=malloc(sizeof(point)*4);

    push_front(org,target,3);print_nD_arr(org);
    push_front(org,target,3);print_nD_arr(org);
    push_front(org,target,3);print_nD_arr(org);
    push_front(org,target1,3);print_nD_arr(org);
    push_front(org,target1,3);print_nD_arr(org);
*/
//  test buliding KD tree //bug fixed//succeed
/*

    node *tree;
    tree=convert_2_KDtree(orgarr,50);//only code for 50, not yet solved other portions!
    print_bt(tree);
    print2DUtil(tree,0);
    */
//test approximate searching k=1
/*

    point target={{31,14,73},0};
    printf("%.1f,%.1f,%.1f\n",target.values[0],target.values[1],target.values[2]);
    point* found=k_nearest_search(1,tree,true,target);//true: approximate search
    print_nD_arr(found);
  */
//test distance correctness--succeed
/*
    //test distance correctness--succeed
    point p1={{3,7,2},0};
    point p2={{12,47,25},0};
    printf("ditance %.1f",distance_calc(p1,p2));
    exit(0);
*/
//test searching k>1-- approximate and back tracking both work
/*

    point target={{14,114,214},0};
    printf("%.1f,%.1f,%.1f\n",target.values[0],target.values[1],target.values[2]);
    point* found=k_nearest_search(5,tree,false,target);//true: approximate search
    print_nD_arr(found);
*/
//test push back
/*

    printf("------------test push\n");
    point target0={{51,32,61},0};
    point target1={{1,32,61},0};
    point* org=malloc(sizeof(point)*4);

    push_back(org,target0,3,false);print_nD_arr(org);
    push_back(org,target0,3,false);print_nD_arr(org);
    push_back(org,target0,3,false);print_nD_arr(org);
    push_back(org,target1,3,false);print_nD_arr(org);
    push_back(org,target1,3,false);print_nD_arr(org);
*/
//test rand_ find_mid_index
/*

    point target={{4.5,104.5,204.5},0};
    int chosen_index;
    chosen_index=find_mid_index(orgarr,target,0);
    printf("point shoud be at %d index",chosen_index);
*/
//build tree with specific portion
/*
    node *tree;
    run_time_debug[0]=clock();
    tree=convert_2_KDtree(orgarr,true);//for testing
    print_bt(tree);
    print2DUtil(tree,0);
    exit(0);
    point target={{14,114,214},0};
    printf("%.1f,%.1f,%.1f\n",target.values[0],target.values[1],target.values[2]);
    point* found=k_nearest_search(5,tree,false,target);//true: approximate search
    print_nD_arr(found);
*/
//These block tested with 4096 points & read write files & approximate precise search with random split KDtree
/*
    //ouput generated point!
//    write_data_to_txt("9points_rand_max39.txt",orgarr);

    //input generated data point
    point* gotarr;
    gotarr=read_data_from_txt("9points_rand_max39.txt");
    show_time("read dmn file");
    print_nD_arr(gotarr);
    show_time("print org arr");
    //test build ran_split KDtree
//    exit(clock()-main_start);
    node *tree;
    tree=convert_2_KDtree(gotarr,true);
    show_time("build tree time spent!");
    print_bt(tree);
    print2DUtil(tree,0);
    show_time("print tree");

    point target={{14,114,214},0};
    printf("%.1f,%.1f,%.1f\n",target.values[0],target.values[1],target.values[2]);
    //test kNN search with precise with k>1
    point* found=k_nearest_search(5,tree,false,target);//true: approximate search false: precise
    show_time("kNN precise");
    print_nD_arr(found);
    show_time("print found");

    printf("%.1f,%.1f,%.1f\n",target.values[0],target.values[1],target.values[2]);
    found=k_nearest_search(5,tree,true,target);//true: approximate search//k=1 as always
    show_time("kNN approximate");
    print_nD_arr(found);
    show_time("print found");
*/
//Traverse KD tree approximately with non-recursive & non-stack just one while loop k=1
/*
    //build tree from disk file

    node* tree;
    tree=convert_2_KDtree(read_data_from_txt("9points_rand_max39.txt"),true);
    print2DUtil(tree,0);
    show_time("tree print Fin");
    point target;
    point* nearest_points;
    int i;
    for(i=0;i<DIM;i++) target.values[i]=((i+1)*(rand()%43))%39;//test target point
    show_time("create_dependency Fin");
    nearest_points=k_nearest_search_wo_recusrion_stack(1,tree,true,target);
    show_time("kNN search fin");

    printf("target: ");print_this_point(target);
    print_nD_arr(nearest_points);
*/

//Generate trees & traverse with gpu
/*
//    point* orgarr=super_gen_rand_arr(8,144);//testing
//    write_data_to_txt("8points_rand_max144.txt",orgarr);//testing
    const int tree_num = 32;
    node** tree;
    hipMallocManaged(&tree,sizeof(node)*tree_num);
    //node* tree[tree_num];
    int i;
    
    point* orgarr=read_data_from_txt("12pow_points_rand_max65535.txt");//NO: 13,14,15,16 ;PNO: 11, 12
    for(i=0;i<tree_num;i++) tree[i]=convert_2_KDtree(orgarr,true);
    free(orgarr);
    //orgprint
    //for(i=0;i<tree_num;i++) {
    //    print2DUtil(tree[i],0);
    //    printf("\n\n------------------------------------------------------\n\n\n\n\n");
    //}
    point target;
    point* found;

    target.values[0] = 33333;
    target.values[1] = 33333;
    target.values[2] = 11111;
    //printf("target point: ");
    //print_this_point_woth(target);
    //printf("\n");

    //GPU part--start!
    hipMallocManaged(&found, sizeof(int) * (tree_num));
    clock_t traverse_start=clock();
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    k_nearest_search_k1_GPU << <20, tree_num >> > (tree, target, tree_num, found);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    //GPU part--END
    printf("ttraverse all trees time taken%3.6f ms\n", time);
    //show four points
    //printf("\n\n==============================\nfound:\n");
    //for(i=0;i<tree_num;i++){
    //    print_this_point(found[i]);
    //    printf("\n");
    //}
    hipFree(found);
    */
// Final. Traverse methods change test.---try query points & tree_num & 1 block & N Threads
 /*
//original
    //int treeloop;
    int tree_num;
    int jmpVal;// skip error mem access
    //for(treeloop=8;treeloop>=7;treeloop--) {//control tree number
        //treeloop = 1;   
        //tree_num = mypow(2, treeloop);//2,4,8,16,32,64,128//,256
        tree_num = 32;
        node **tree;
        //hipMallocManaged(&tree, sizeof(node) * max_tree_num);
        
        unsigned long long i;
        static unsigned long long rectify = 0;
        int j,previous_j,k;
        point *orgarr;
        

        unsigned long long queries_max = mypow(2, 14);//14
        unsigned long long queries_min = mypow(2, 2);
        unsigned long long loop_interval,queries_interval = mypow(2, 9);//9
        loop_interval = queries_interval;
        for (i = queries_max; i >= queries_min; i -= loop_interval) {//control query points number
            
            orgarr = super_gen_rand_arr(i, 65535);//generated immediately
            



            //orgarr = super_gen_seq_arr(i, false);
           
            hipMallocManaged(&tree, sizeof(node) * (tree_num));
            for (j = 0; j < tree_num; j++) {
                jmpVal = setjmp(jmpbuffer);
                if (jmpVal == 0) {//norm case
                    

                    tree[j] = convert_2_KDtree(orgarr, true, rectify);
                    rectify = 0;
                    previous_j= j;

                }
                else if (jmpVal == 1) {// Null is return by CudaMallocManagement occur, skip, becasue excess Cuda ctrl's memory 
                    printf("newNodes is NULL!\n");
                    jmpVal = 0;
                    //j = store_j;
                    continue;
                }
                else if (jmpVal == 2) {//if allocated node memory insufficient
                    rectify++;
                    jmpVal = 0;
                    j = previous_j;//j->j-1
                    free(tree[j]);
                }

            }
            free(orgarr);
            //printf("traverse------------------------------------------------\n");
            point target;
            point* found;

            target.values[0] = 32751;
            target.values[1] = 33751;
            target.values[2] = 30000;
            //target.values[0] = 4096;
            //target.values[1] = 4196;
            //target.values[2] = 4296;
            //printf("target point: ");
            //print_this_point_woth(target);
            //printf("\nGPU part--start!\n");


            hipMallocManaged(&found, sizeof(point) * (tree_num));//was sizeof(int)
            clock_t traverse_start = clock();
            float time;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
            k_nearest_search_k1_GPU << < 1, tree_num >> > (tree, target, tree_num, found);
            //k_nearest_search_k1_GPU << < tree_num, 32 >> > (tree, target, tree_num, found);
            hipDeviceSynchronize();//CPU stop

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            //GPU part--END

            printf("ttraverse all trees time taken%3.6f s\n", time);
            //show four points
            //printf("found:\n");
            //for (j = 0; j < tree_num; j++) {
            //    printf("%d\t", j);
            //    print_this_point(found[j]);
            //    printf("\n");
            //}


            //traverse---END
            hipFree(found);//need
            //for (j = 0; j < tree_num; j++) {
            //    hipFree(tree[j]);
            //}
            hipFree(tree);//need
            //free(orgarr);
        //write text to disk here
            printf("\n\n\ntree number %d\tquery points number %llu, true interval %llu \n", tree_num, i,loop_interval);
            write_traverseresult_to_disk(tree_num, i, time,queries_max,queries_min,queries_interval);
            if ((i < 4 * loop_interval)&&(loop_interval!=1)) loop_interval /= 2;
        }

    */
//Final. Generate query points to compare performance in the future.
/*
    unsigned long long i;
    point* orgarr;
    unsigned long long queries_max = mypow(2, 14);//14
    unsigned long long queries_min = mypow(2, 2);
    unsigned long long loop_interval, queries_interval = mypow(2, 9);//9
    
    //write text
    char fname_format[] = "rand_queries_%llu.txt";
    char fname[sizeof fname_format + 128];
    


    loop_interval = queries_interval;

    for (i = queries_max; i >= queries_min; i -= loop_interval) {//control query points number
        orgarr = super_gen_rand_arr(i, 65535);//generated immediately
        sprintf(fname, fname_format, i);
        write_data_to_txt(fname, orgarr);
        if ((i < 4 * loop_interval) && (loop_interval != 1)) loop_interval /= 2;
        printf(".");
    }
    */
// Final. Traverse metods test; input data from disk. ->1 block  tree_num threads
/*
    int tree_num;
    int jmpVal;// skip error mem access
    tree_num = 32;
    node** tree;
    unsigned long long i;
    static unsigned long long rectify = 0;
    int j, previous_j, k;
    point* orgarr;
    unsigned long long queries_max = mypow(2, 14);//14
    unsigned long long queries_min = mypow(2, 2);
    unsigned long long loop_interval, queries_interval = mypow(2, 9);//9
    loop_interval = queries_interval;
    char fname_format[] = "rand_queries_%llu.txt";
    char fname[sizeof fname_format + 128];
    for (i = queries_max; i >= queries_min; i -= loop_interval) {//control query points number
        sprintf(fname, fname_format, i);
        orgarr = read_data_from_txt(fname);

        hipMallocManaged(&tree, sizeof(node) * (tree_num));
        for (j = 0; j < tree_num; j++) {
            jmpVal = setjmp(jmpbuffer);
            if (jmpVal == 0) {//norm case
                tree[j] = convert_2_KDtree(orgarr, true, rectify);
                rectify = 0;
                previous_j = j;
            }
            else if (jmpVal == 1) {// Null is return by CudaMallocManagement occur, skip, becasue excess Cuda ctrl's memory 
                printf("newNodes is NULL!\n");
                jmpVal = 0;
                //j = store_j;
                continue;
            }
            else if (jmpVal == 2) {//if allocated node memory insufficient
                rectify++;
                jmpVal = 0;
                j = previous_j;//j->j-1
            }
        }
        free(orgarr);
        //printf("traverse------------------------------------------------\n");
        point target;
        point* found;
        target.values[0] = 32751;
        target.values[1] = 33751;
        target.values[2] = 30000;
         hipMallocManaged(&found, sizeof(point) * (tree_num));//was sizeof(int)
        clock_t traverse_start = clock();
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        k_nearest_search_k1_GPU << < 1, tree_num >> > (tree, target, tree_num, found);
        hipDeviceSynchronize();//CPU stop
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        printf("ttraverse all trees time taken%3.6f s\n", time);
        hipFree(found);//need
        hipFree(tree);//need
        //free(orgarr);
        //write text to disk here
        printf("\n\n\ntree number %d\tquery points number %llu, true interval %llu \n", tree_num, i, loop_interval);
        write_traverseresult_to_disk(tree_num, i, time, queries_max, queries_min, queries_interval);
        if ((i < 4 * loop_interval) && (loop_interval != 1)) loop_interval /= 2;
    }
    */

 // Final. Traverse metods test; input data from disk. ->tree_num block 1 thread for each block
    /*
    int tree_num;
    int jmpVal;// skip error mem access
    tree_num = 32;
    node** tree;
    unsigned long long i;
    static unsigned long long rectify = 0;
    int j, previous_j, k;
    point* orgarr;
    unsigned long long queries_max = mypow(2, 14);//14
    unsigned long long queries_min = mypow(2, 2);
    unsigned long long loop_interval, queries_interval = mypow(2, 2);
    loop_interval = queries_interval;
    char fname_format[] = "rand_queries_%llu.txt";
    char fname[sizeof fname_format + 128];
    write_Data_head = true;
    for (i = queries_max; i >= queries_min; i /= loop_interval) {//control query points number
        sprintf(fname, fname_format, i);
        orgarr = read_data_from_txt(fname);

        hipMallocManaged(&tree, sizeof(node) * (tree_num));
        for (j = 0; j < tree_num; j++) {
            jmpVal = setjmp(jmpbuffer);
            if (jmpVal == 0) {//norm case
                tree[j] = convert_2_KDtree(orgarr, true, rectify);
                rectify = 0;
                previous_j = j;
            }
            else if (jmpVal == 1) {// Null is return by CudaMallocManagement occur, skip, becasue excess Cuda ctrl's memory 
                printf("newNodes is NULL!\n");
                jmpVal = 0;
                //j = store_j;
                continue;
            }
            else if (jmpVal == 2) {//if allocated node memory insufficient
                rectify++;
                jmpVal = 0;
                j = previous_j;//j->j-1
            }
        }
        free(orgarr);
        //printf("traverse------------------------------------------------\n");
        point target;
        point* found;
        target.values[0] = 32751;
        target.values[1] = 33751;
        target.values[2] = 30000;
        hipMallocManaged(&found, sizeof(point) * (tree_num));//was sizeof(int)
        float time;
        hipEvent_t start, stop;
        int block_num,thread_num_per_block;
        
        for (j = 1; j <= tree_num; j++) {
            block_num = j;
            printf("block_num%d\n", block_num);
            
            thread_num_per_block = tree_num/block_num+(int)(tree_num % block_num!=0);
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
            k_nearest_search_k1_GPU << < block_num, thread_num_per_block >> > (tree, target, tree_num, found);
            hipDeviceSynchronize();//CPU stop
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            printf("ttraverse all trees time taken%3.6f s\n", time);
            hipFree(found);//need
            hipFree(tree);//need
            //free(orgarr);
            //write text to disk here
            printf("tree number %d\tquery points number %llu, true interval %llu \n\n\n", tree_num, i, loop_interval);
            write_traverseresult_to_disk(tree_num, i, time, queries_max, queries_min, queries_interval,block_num,thread_num_per_block);
            //exit(322);
        }
        if ((i < 4 * loop_interval) && (loop_interval != 1)) loop_interval /= 2;
        else if (loop_interval == 1) break;
    }
    
    */

   // change tree order

    int tree_num;
    int jmpVal;// skip error mem access
    tree_num = 32;
    node** tree;
    node** tree_reversed;
    unsigned long long i;
    static unsigned long long rectify = 0;
    int j, previous_j, k;
    point* orgarr;
    unsigned long long queries_max = mypow(2, 14);//14
    unsigned long long queries_min = mypow(2, 2);
    unsigned long long loop_interval, queries_interval = mypow(2, 2);
    loop_interval = queries_interval;
    char fname_format[] = "rand_queries_%llu.txt";
    char fname[sizeof fname_format + 128];
    write_Data_head = true;
    for (i = queries_max; i >= queries_min; i /= loop_interval) {//control query points number
        sprintf(fname, fname_format, i);
        orgarr = read_data_from_txt(fname);

        hipMallocManaged(&tree, sizeof(node) * (tree_num));
        hipMallocManaged(&tree_reversed, sizeof(node) * (tree_num));
        for (j = 0; j < tree_num; j++) {
            jmpVal = setjmp(jmpbuffer);
            if (jmpVal == 0) {//norm case
                tree[j] = convert_2_KDtree(orgarr, true, rectify);
                tree_reversed[tree_num - j] = tree[j];
                rectify = 0;
                previous_j = j;
            }
            else if (jmpVal == 1) {// Null is return by CudaMallocManagement occur, skip, becasue excess Cuda ctrl's memory 
                printf("newNodes is NULL!\n");
                jmpVal = 0;
                //j = store_j;
                continue;
            }
            else if (jmpVal == 2) {//if allocated node memory insufficient
                rectify++;
                jmpVal = 0;
                j = previous_j;//j->j-1
            }
        }
        free(orgarr);
        //printf("traverse------------------------------------------------\n");
        point target;
        point* found;
        target.values[0] = 32751;
        target.values[1] = 33751;
        target.values[2] = 30000;
        hipMallocManaged(&found, sizeof(point) * (tree_num));//was sizeof(int)
        float time;
        hipEvent_t start, stop;
        int block_num, thread_num_per_block;

        for (j = 1; j <= tree_num; j++) {
            block_num = j;
            printf("block_num%d\n", block_num);

            thread_num_per_block = tree_num / block_num + (int)(tree_num % block_num != 0);
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
            k_nearest_search_k1_GPU << < block_num, thread_num_per_block >> > (tree, target, tree_num, found);
            hipDeviceSynchronize();//CPU stop
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            printf("ttraverse all trees time taken%3.6f s\n", time);
            //hipFree(found);//need
            //hipFree(tree);//need
            //free(orgarr);
            //write text to disk here
            printf("tree number %d\tquery points number %llu, true interval %llu \n\n\n", tree_num, i, loop_interval);
            write_traverseresult_to_disk('n',tree_num, i, time, queries_max, queries_min, queries_interval, block_num, thread_num_per_block);
        }
        hipFree(tree);//need
        ////make reversed
        //node* tmp;
        //for (j = 0; j <= tree_num/2; j++) {
        //    tmp = tree[j];
        //    tree[j] = tree[tree_num - j];
        //    tree[tree_num - j] = tmp;
        //}

        //reverse tree traverse
        for (j = 1; j <= tree_num; j++) {
            block_num = j;
            printf("block_num%d\n", block_num);

            thread_num_per_block = tree_num / block_num + (int)(tree_num % block_num != 0);
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
            k_nearest_search_k1_GPU << < block_num, thread_num_per_block >> > (tree_reversed, target, tree_num, found);
            hipDeviceSynchronize();//CPU stop
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            printf("ttraverse all trees time taken%3.6f s\n", time);
            //hipFree(found);//need
            //hipFree(tree);//need
            //free(orgarr);
            //write text to disk here
            printf("tree number %d\tquery points number %llu, true interval %llu \n\n\n", tree_num, i, loop_interval);
            write_traverseresult_to_disk('r',tree_num, i, time, queries_max, queries_min, queries_interval, block_num, thread_num_per_block);

        }
        hipFree(found);//need
        
        hipFree(tree_reversed);//need
        if ((i < 4 * loop_interval) && (loop_interval != 1)) loop_interval /= 2;
        else if (loop_interval == 1) break;
    }
   
    return clock()-main_start;
}